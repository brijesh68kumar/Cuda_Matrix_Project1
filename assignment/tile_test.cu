#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
// Thread block sizes
#define BLOCK_SIZE 4096
#define TILE_WIDTH 4
// Matrix dimensions
// (chosen as multiples of the thread block size for simplicity)
#define MATRIX_SIZE 1* BLOCK_SIZE
#define WA (MATRIX_SIZE) // Matrix A width
#define HA (MATRIX_SIZE) // Matrix A height
#define WB (MATRIX_SIZE) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height
// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
void printOutput(float *C, char a){
    int i=0;
    printf("Printing %c\n", a);
    for(i=0;i<100;i++){
        printf("%f\t", *C++);
    }
}
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
   // Calculate the row index of the Pd element and M
   int Row = blockIdx.y*BLOCK_SIZE + threadIdx.y;
   // Calculate the column idenx of Pd and N
   int Col = blockIdx.x*BLOCK_SIZE + threadIdx.x;
   float Pvalue = 0;
   // each thread computes one element of the block sub-matrix
   for (int k = 0; k < Width; ++k)
       Pvalue += Md[Row*Width+k] * Nd[k*Width+Col];
   Pd[Row*Width+Col] = Pvalue;
}
__global__ void MatrixMulKernelTiled(float* Md, float* Nd, float* Pd, int Width)
{
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    // Identify the row and column of the Pd element to work on
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    float Pvalue = 0;
    // Loop over the Md and Nd tiles required to compute the Pd element
    for (int m = 0; m < Width/TILE_WIDTH; ++m) {
    // Collaborative loading of Md and Nd tiles into shared memory
        Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)];
        Nds[ty][tx] = Nd[Col + (m*TILE_WIDTH + ty)*Width];
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += Mds[ty][k] * Nds[k][tx];
        __syncthreads();
    }
    Pd[Row*Width+Col] = Pvalue;
}
void MatrixMulOnHost(float* M, float* N, float* P, int Width)
{
    for (int i = 0; i < Width; ++i)
        for (int j = 0; j < Width; ++j) {
            double sum = 0;
            for (int k = 0; k < Width; ++k) {
                double a = M[i * Width + k];
                double b = N[k * Width + j];
                sum += a * b;
            }
            P[i * Width + j] = sum;
        }
}
float runMatrixWithOutShared(float *h_A, float *h_B, unsigned int mem_size_A, unsigned int mem_size_B)
{
    hipEvent_t start, stop;
    hipEvent_t startCopyHtoD, startCopyDtoH, stopCopyHtoD, stopCopyDtoH;
    char a='a',b='b',c='c';
    printOutput(h_A,a);
    printOutput(h_B,b);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startCopyHtoD);
    hipEventCreate(&stopCopyHtoD);
    hipEventCreate(&startCopyDtoH);
    hipEventCreate(&stopCopyDtoH);
    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);
    hipEventRecord(startCopyHtoD);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,hipMemcpyHostToDevice);
    hipEventRecord(stopCopyHtoD);
    hipEventSynchronize(stopCopyHtoD);
    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);
    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);
    
    hipEventRecord(start);
    // execute the kernel
    MatrixMulKernel<<< grid, threads >>>(d_A, d_B, d_C, WB);
    hipEventRecord(stop);
    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    hipEventRecord(startCopyDtoH);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,hipMemcpyDeviceToHost);
    hipEventRecord(stopCopyDtoH);
    hipEventSynchronize(stop);
    hipEventSynchronize(stopCopyDtoH);
    printOutput(h_C,c);
    float copyHtoD = 0;
    float copyDtoH = 0;
    float kernelRunTime = 0;
    hipEventElapsedTime(&copyDtoH, startCopyDtoH, stopCopyDtoH);
    hipEventElapsedTime(&copyHtoD, startCopyHtoD, stopCopyHtoD);
    hipEventElapsedTime(&kernelRunTime, start, stop);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    printf("Copy Time From H To D: %f\n",copyHtoD);
    printf("Copy Time From D To H: %f\n",copyDtoH);
    return kernelRunTime;
}
float runMatrixWithShared(float *h_A, float *h_B, unsigned int mem_size_A, unsigned int mem_size_B){
    hipEvent_t start, stop;
    hipEvent_t startCopyHtoD, startCopyDtoH, stopCopyHtoD, stopCopyDtoH;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startCopyHtoD);
    hipEventCreate(&stopCopyHtoD);
    hipEventCreate(&startCopyDtoH);
    hipEventCreate(&stopCopyDtoH);
    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);
    hipEventRecord(startCopyHtoD);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,hipMemcpyHostToDevice) ;
    hipMemcpy(d_B, h_B, mem_size_B,hipMemcpyHostToDevice);
    
    hipEventRecord(stopCopyHtoD);
    hipEventSynchronize(stopCopyHtoD);
    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);
    dim3 dimThreads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(WC / dimThreads.x, HA / dimThreads.y);
    hipEventRecord(start);
    MatrixMulKernelTiled<<<dimGrid, dimThreads>>>(d_A, d_B, d_C,WB);
    hipEventRecord(stop);
    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    hipEventRecord(startCopyDtoH);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,hipMemcpyDeviceToHost);
    hipEventRecord(stopCopyDtoH);
    hipEventSynchronize(stop);
    hipEventSynchronize(stopCopyDtoH);
    char c = 'c';
    printOutput(h_C,c);
    float copyHtoD = 0;
    float copyDtoH = 0;
    float kernelRunTime = 0;
    hipEventElapsedTime(&copyDtoH, startCopyDtoH, stopCopyDtoH);
    hipEventElapsedTime(&copyHtoD, startCopyHtoD, stopCopyHtoD);
    hipEventElapsedTime(&kernelRunTime, start, stop);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    printf("Copy Time From H To D: %f\n",copyHtoD);
    printf("Copy Time From D To H: %f\n",copyDtoH);
    return kernelRunTime;
}
int main()
{
	/*long long ctime,cudatime,cudatiletime,hosttime,inittime,totaltime;
	struct timeval stime,stime2,etime, etime2,initstime,initetime,totalstime,totaletime,tilestime,tileetime;
    gettimeofday(&totalstime,0);
    gettimeofday(&initstime,0);
*/
    printf("Matrix Size =%dX%d \n",MATRIX_SIZE,MATRIX_SIZE);
    printf("Tile Width = %dX%d\n",TILE_WIDTH,TILE_WIDTH);
    hipEvent_t startInit, stopInit;
    hipEventCreate(&startInit);
    hipEventCreate(&stopInit);
    hipEventRecord(startInit);
    srand(2006);
    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    hipEventRecord(stopInit);
    float initTime = 0;
    hipEventElapsedTime(&initTime, startInit, stopInit);
    printf("Init Time: %f\n", initTime);
/*----------------PARALLEL EXECUTION BEGINS HERE ----------------------------*/
    
    printf("Starting Without Shared Memory\n\n");
    float matTime = runMatrixWithOutShared(h_A,h_B,mem_size_A,mem_size_B);
    printf("Mat Time No TILING: %f\n\n", matTime);
/*------------------PARALLEL EXECUTION ENDS HERE ----------------------------*/    
    
/* -----------------TILING EXECUTION BEGINS HERE-----------------------------*/
    
    printf("Starting With Shared Memory\n\n");
    float matTimeWithTile = runMatrixWithShared(h_A,h_B,mem_size_A,mem_size_B);
    printf("Mat Time With TILING: %f\n\n", matTimeWithTile);
/*------------------------------ TILING ENDS HERE ---------------------------------*/
/*
    gettimeofday(&stime,0);
    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    MatrixMulOnHost(h_A, h_B, reference, WB);
	gettimeofday(&etime,0);
    hosttime = (etime.tv_sec-stime.tv_sec)*1000000LL + etime.tv_usec-stime.tv_usec;
	//printf("host: %lld\ncuda: %lld\ncuda, w/copy: %lld\n", hosttime, cudatime, ctime);
    gettimeofday(&totaletime,0);
    totaltime = (totaletime.tv_sec-totalstime.tv_sec)*1000000LL + totaletime.tv_usec-totalstime.tv_usec;
    */
    // clean up memory
    free(h_A);
    free(h_B);
    //free(h_C);
    //free(reference);
    //cudaFree(d_A);
    //cudaFree(d_B);
    //cudaFree(d_C);
    //cudaThreadExit();
   /* printf("Total Time: %lld\n",totaltime );    
    printf("Initialization Time: %lld\n",inittime);
    printf("Copy Time: %lld\n",ctime-cudatime);
    printf("Parallel Time: %lld\n",cudatime);
    printf("Host Time: %lld\n", hosttime);
    printf("Tile Parallel Time: %lld\n",cudatiletime);
    printf("Sum %lld\n", inittime+ctime+hosttime+cudatiletime);*/
}

