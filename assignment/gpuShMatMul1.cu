
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

/* change dimension size as needed */
const int dimension = 4096 ;
const int blocksize = 64;
const int K = 4;
const int tilewidth = 2 ;


struct timeval tv; 

__global__ void gpuSmMM( float *Ad , float *Bd , float *Cd , int dimention )
{

        //Taking shared array to break the MAtrix in Tile widht and fatch them in that array per ele
          __shared__ float Ads [tilewidth][tilewidth] ;
          __shared__ float Bds [tilewidth][tilewidth] ;
         // calculate thread id
          unsigned int col = tilewidth*blockIdx.x + threadIdx.x ;
          unsigned int row = tilewidth*blockIdx.y + threadIdx.y ;
        for (int m = 0 ; m<dimention/tilewidth ; m++ ) // m indicate number of phase
			{
            Ads[threadIdx.y][threadIdx.x] =  Ad[row*dimention + (m*tilewidth + threadIdx.x)]  ;
            Bds[threadIdx.y][threadIdx.x] =  Bd[ ( m*tilewidth + threadIdx.y) * dimention + col] ;
         __syncthreads() ; // for syncronizeing the threads
         // Do for tile
				for ( int k1 = 0; k1<tilewidth ; k1++ )
                       Cd[row*dimention + col]+= Ads[threadIdx.x][k1] * Bds[k1][threadIdx.y] ;
         __syncthreads() ; // for syncronizeing the threads

			}
}


int main(int argc, char *argv[])
{
hipEvent_t start_i, stop_i,start_mc_h2d, stop_mc_h2d,start_mc_d2h, stop_mc_d2h,start_pl, stop_pl;
		float time_i,time_mc_h2d,time_mc_d2h,time_pl;

		hipEventCreate(&start_i);
		hipEventCreate(&stop_i);
		
		hipEventCreate(&start_mc_h2d);
		hipEventCreate(&stop_mc_h2d);

		hipEventCreate(&start_mc_d2h);
		hipEventCreate(&stop_mc_d2h);
		
		hipEventCreate(&start_pl);
		hipEventCreate(&stop_pl);
		
		int i, j;
        float *A, *B, *C;// start, end;
		float *Ad, *Bd, *Cd;

		hipEventRecord( start_i, 0 );
		
        A = (float*)malloc(dimension*dimension*sizeof(float));
        B = (float*)malloc(dimension*dimension*sizeof(float));
        C = (float*)malloc(dimension*dimension*sizeof(float));		
		
        srand(292);



        for(i = 0; i < dimension; i++)
                for(j = 0; j < dimension; j++)
                {   
                        A[dimension*i+j] = (rand()/(RAND_MAX + 1.0));
                        B[dimension*i+j] = (rand()/(RAND_MAX + 1.0));
                        C[dimension*i+j] = 0.0;
                }   
				
		hipEventRecord( stop_i, 0 );
		hipEventSynchronize( stop_i );
		
		hipEventElapsedTime( &time_i, start_i, stop_i );


		hipEventRecord( start_mc_h2d, 0 );
		
		hipMalloc( (void**)&Ad, dimension*dimension*sizeof(float) );
		hipMemcpy( Ad, A, dimension*dimension*sizeof(float), hipMemcpyHostToDevice );

		hipMalloc( (void**)&Bd, dimension*dimension*sizeof(float) );
		hipMemcpy( Bd, B, dimension*dimension*sizeof(float), hipMemcpyHostToDevice );
		
		hipMalloc( (void**)&Cd, dimension*dimension*sizeof(float) );

		hipEventRecord( stop_mc_h2d, 0 );
		hipEventSynchronize( stop_mc_h2d );
		
		hipEventElapsedTime( &time_mc_h2d, start_mc_h2d, stop_mc_h2d );

		

        //start = timestamp();

		hipEventRecord( start_pl, 0 );
		dim3 threadBlock(blocksize,blocksize);
		dim3 grid(K,K);
		
		gpuSmMM<<<grid,threadBlock>>>( Ad,Bd,Cd,dimension);

        //end = timestamp();

		hipEventRecord( stop_pl, 0 );
		hipEventSynchronize( stop_pl );
		
		hipEventElapsedTime( &time_pl, start_pl, stop_pl );

		hipEventRecord( start_mc_d2h, 0 );
		hipMemcpy(C,Cd,dimension*dimension*sizeof(float),hipMemcpyDeviceToHost);
		hipEventRecord( stop_mc_d2h, 0 );
		hipEventSynchronize( stop_mc_d2h );
		
		hipEventElapsedTime( &time_mc_d2h, start_mc_d2h, stop_mc_d2h );

		
		//printf("IT: %f ", time_i);
		printf("MC: %f ", ( time_mc_d2h + time_mc_h2d ) );
		printf("PLT: %f \n ", time_pl);
		//printf("T:%f .... \n\n", (time_pl + time_mc_d2h + time_mc_h2d+time_i));

		
        //printf("\nsecs:%f\n", end-start);

		hipEventDestroy( start_i );
		hipEventDestroy( stop_i );

		hipEventDestroy( start_mc_d2h );
		hipEventDestroy( stop_mc_d2h );

		hipEventDestroy( start_mc_h2d );
		hipEventDestroy( stop_mc_h2d );
		
		hipEventDestroy( start_pl );
		hipEventDestroy( stop_pl );
		
        free(A);
        free(B);
        free(C);
		hipFree(Ad);
		hipFree(Bd);
		hipFree(Cd);

        return 0;
}
