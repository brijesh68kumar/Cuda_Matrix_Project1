
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

/* change dimension size as needed */
const int dimension = 4096 ;
const int blocksize = 10;
const int K = 1;


struct timeval tv; 

__global__ void gpuMM(float *A, float *B, float *C, int N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	float sum = 0.f;
	for (int n = 0; n < N; ++n)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}


int main(int argc, char *argv[])
{
		hipEvent_t start_i, stop_i,start_mc_h2d, stop_mc_h2d,start_mc_d2h, stop_mc_d2h,start_pl, stop_pl;
		float time_i,time_mc_h2d,time_mc_d2h,time_pl;

		hipEventCreate(&start_i);
		hipEventCreate(&stop_i);
		
		hipEventCreate(&start_mc_h2d);
		hipEventCreate(&stop_mc_h2d);

		hipEventCreate(&start_mc_d2h);
		hipEventCreate(&stop_mc_d2h);
		
		hipEventCreate(&start_pl);
		hipEventCreate(&stop_pl);
		
		int i, j;
        float *A, *B, *C;// start, end;
		float *Ad, *Bd, *Cd;

		hipEventRecord( start_i, 0 );
		
        A = (float*)malloc(dimension*dimension*sizeof(float));
        B = (float*)malloc(dimension*dimension*sizeof(float));
        C = (float*)malloc(dimension*dimension*sizeof(float));		
		
        srand(292);



        for(i = 0; i < dimension; i++)
                for(j = 0; j < dimension; j++)
                {   
                        A[dimension*i+j] = (rand()/(RAND_MAX + 1.0));
                        B[dimension*i+j] = (rand()/(RAND_MAX + 1.0));
                        C[dimension*i+j] = 0.0;
                }   
				
		hipEventRecord( stop_i, 0 );
		hipEventSynchronize( stop_i );
		
		hipEventElapsedTime( &time_i, start_i, stop_i );


		hipEventRecord( start_mc_h2d, 0 );
		
		hipMalloc( (void**)&Ad, dimension*dimension*sizeof(float) );
		hipMemcpy( Ad, A, dimension*dimension*sizeof(float), hipMemcpyHostToDevice );

		hipMalloc( (void**)&Bd, dimension*dimension*sizeof(float) );
		hipMemcpy( Bd, B, dimension*dimension*sizeof(float), hipMemcpyHostToDevice );
		
		hipMalloc( (void**)&Cd, dimension*dimension*sizeof(float) );

		hipEventRecord( stop_mc_h2d, 0 );
		hipEventSynchronize( stop_mc_h2d );
		
		hipEventElapsedTime( &time_mc_h2d, start_mc_h2d, stop_mc_h2d );

		

        //start = timestamp();

		hipEventRecord( start_pl, 0 );
		dim3 threadBlock(blocksize,blocksize);
		dim3 grid(K,K);
		
		gpuMM<<<grid,threadBlock>>>( Ad,Bd,Cd,dimension);

        //end = timestamp();

		hipEventRecord( stop_pl, 0 );
		hipEventSynchronize( stop_pl );
		
		hipEventElapsedTime( &time_pl, start_pl, stop_pl );

		hipEventRecord( start_mc_d2h, 0 );
		hipMemcpy(C,Cd,dimension*dimension*sizeof(float),hipMemcpyDeviceToHost);
		hipEventRecord( stop_mc_d2h, 0 );
		hipEventSynchronize( stop_mc_d2h );
		
		hipEventElapsedTime( &time_mc_d2h, start_mc_d2h, stop_mc_d2h );

		
		printf("\n IT : %f   ", time_i);
		printf(" MCT : %f", ( time_mc_d2h + time_mc_h2d ) );
		printf(" PLT:%f ", time_pl);
		printf(" Total:%f.... \n\n", (time_pl + time_mc_d2h + time_mc_h2d+time_i));

		
        //printf("\nsecs:%f\n", end-start);

		hipEventDestroy( start_i );
		hipEventDestroy( stop_i );

		hipEventDestroy( start_mc_d2h );
		hipEventDestroy( stop_mc_d2h );

		hipEventDestroy( start_mc_h2d );
		hipEventDestroy( stop_mc_h2d );
		
		hipEventDestroy( start_pl );
		hipEventDestroy( stop_pl );
		
        free(A);
        free(B);
        free(C);
		hipFree(Ad);
		hipFree(Bd);
		hipFree(Cd);

        return 0;
}
