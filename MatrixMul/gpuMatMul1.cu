
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

/* change dimension size as needed */
const int dimension = 512 ;
const int blocksize = 32;
const int K = 1;


struct timeval tv; 

__global__ void gpuMM(float *A, float *B, float *C, int N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	float sum = 0.f;
	for (int n = 0; n < N; ++n)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}


/*float timestamp()
{
        float t;
        gettimeofday(&tv, NULL);
        t = tv.tv_sec + (tv.tv_usec/1000000.0);
        return t;
}
*/

int main(int argc, char *argv[])
{
		hipEvent_t start1, stop1;
		float time;

		hipEventCreate(&start1);
		hipEventCreate(&stop1);
        
		int i, j;
        float *A, *B, *C;// start, end;
		float *Ad, *Bd, *Cd;

        A = (float*)malloc(dimension*dimension*sizeof(float));
        B = (float*)malloc(dimension*dimension*sizeof(float));
        C = (float*)malloc(dimension*dimension*sizeof(float));		
		
        srand(292);

        for(i = 0; i < dimension; i++)
                for(j = 0; j < dimension; j++)
                {   
                        A[dimension*i+j] = (rand()/(RAND_MAX + 1.0));
                        B[dimension*i+j] = (rand()/(RAND_MAX + 1.0));
                        C[dimension*i+j] = 0.0;
                }   
				
		hipMalloc( (void**)&Ad, dimension*dimension*sizeof(float) );
		hipMemcpy( Ad, A, dimension*dimension*sizeof(float), hipMemcpyHostToDevice );

		hipMalloc( (void**)&Bd, dimension*dimension*sizeof(float) );
		hipMemcpy( Bd, B, dimension*dimension*sizeof(float), hipMemcpyHostToDevice );
		
		hipMalloc( (void**)&Cd, dimension*dimension*sizeof(float) );

		dim3 threadBlock(blocksize,blocksize);
		dim3 grid(K,K);
		
        //start = timestamp();
		hipEventRecord( start1, 0 );

		gpuMM<<<grid,threadBlock>>>( Ad,Bd,Cd,dimension);

        //end = timestamp();

		hipEventRecord( stop1, 0 );
		hipEventSynchronize( stop1 );
		
		hipEventElapsedTime( &time, start1, stop1 );

		printf("\nsecs:%f\n", time);
		hipEventDestroy( start1 );
		hipEventDestroy( stop1 );

		hipMemcpy(C,Cd,dimension*dimension*sizeof(float),hipMemcpyDeviceToHost);

        //printf("\nsecs:%f\n", end-start);

        free(A);
        free(B);
        free(C);
		hipFree(Ad);
		hipFree(Bd);
		hipFree(Cd);

        return 0;
}
