
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

/* change dimension size as needed */
const int dimension = 32 ;
struct timeval tv; 

double timestamp()
{
        double t;
        gettimeofday(&tv, NULL);
        t = tv.tv_sec + (tv.tv_usec/1000000.0);
        return t;
}

int main(int argc, char *argv[])
{
	hipEvent_t start1, stop1;
	float time;

	hipEventCreate(&start1);
	hipEventCreate(&stop1);

        int i, j, k;
        double *A, *B, *C; 
	double  start, end;

        A = (double*)malloc(dimension*dimension*sizeof(double));
        B = (double*)malloc(dimension*dimension*sizeof(double));
        C = (double*)malloc(dimension*dimension*sizeof(double));

        srand(292);

        for(i = 0; i < dimension; i++)
                for(j = 0; j < dimension; j++)
                {   
                        A[dimension*i+j] = (rand()/(RAND_MAX + 1.0));
                        B[dimension*i+j] = (rand()/(RAND_MAX + 1.0));
                        C[dimension*i+j] = 0.0;
                }   

	hipEventRecord( start1, 0 );


        start = timestamp();
        for(i = 0; i < dimension; i++)
                for(j = 0; j < dimension; j++)
                        for(k = 0; k < dimension; k++)
                                C[dimension*i+j] += A[dimension*i+k] *
                                        B[dimension*k+j];

        end = timestamp();
		
	hipEventRecord( stop1, 0 );
	hipEventSynchronize( stop1 );
	
	hipEventElapsedTime( &time, start1, stop1 );
	printf("\nsecs:%f\n", time ) ; // cudaEventElapsedTime( &time, start1, stop1 ));
	hipEventDestroy( start1 );
	hipEventDestroy( stop1 );

        printf("\nsecs:%f\n", end-start);

        free(A);
        free(B);
        free(C);

        return 0;

}
