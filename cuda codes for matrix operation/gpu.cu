
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <iostream>

using namespace std;


const int dimension = 1024;
const int blocksize = 32;
const int K = 1;


struct timeval tv;

__global__ void gpuMM(float *A, float *B, float *C, int N)
{
        // Matrix multiplication for NxN matrices C=A*B
        // Each thread computes a single element of C
        int row = blockIdx.y*blockDim.y + threadIdx.y;
        int col = blockIdx.x*blockDim.x + threadIdx.x;

        float sum = 0.0;
        for (int n = 0; n < N; ++n)
            sum += A[row*N+n]*B[n*N+col];

        C[row*N+col] = sum;

//	if(row%50 ==5)
//		printf("%f \t %f \t %f\n",A[row*N+col], B[row*N+col], C[row*N+col]);

}


int main(int argc, char *argv[])
{
                hipEvent_t start1, stop1;
                float time;

                hipEventCreate(&start1);
                hipEventCreate(&stop1);

                int i, j;
	        float *A, *B, *C;// start, end;
                float *Ad, *Bd, *Cd;

        A = (float*)malloc(dimension*dimension*sizeof(float));
        B = (float*)malloc(dimension*dimension*sizeof(float));
        C = (float*)malloc(dimension*dimension*sizeof(float));

        srand(292);

        for(i = 0; i < dimension; i++)
                for(j = 0; j < dimension; j++)
                {
                        A[dimension*i+j] = 1;
                        B[dimension*i+j] = 2;
                        C[dimension*i+j] = 0.0;
                }

                hipMalloc( (void**)&Ad, dimension*dimension*sizeof(float) );
                hipMemcpy( Ad, A, dimension*dimension*sizeof(float), hipMemcpyHostToDevice );

                hipMalloc( (void**)&Bd, dimension*dimension*sizeof(float) );
                hipMemcpy( Bd, B, dimension*dimension*sizeof(float), hipMemcpyHostToDevice );

                hipMalloc( (void**)&Cd, dimension*dimension*sizeof(float) );

                dim3 threadBlock(blocksize,blocksize);
                dim3 grid(K,K);

                hipEventRecord( start1, 0 );

                gpuMM<<<grid,threadBlock>>>( Ad,Bd,Cd,dimension);

                hipEventRecord( stop1, 0 );
                hipEventSynchronize( stop1 );

                hipEventElapsedTime( &time, start1, stop1 );
                hipMemcpy( C, Cd, dimension*dimension*sizeof(float), hipMemcpyDeviceToHost );


        for(i = 0; i < 20; i++)
	    {
        	{
        		printf("%f   ",C[i]);
    		}
		if(i%10 ==9)
        	    printf("\n");
	    }



                printf("\ntime taken:%f  sec\n", time);
                hipEventDestroy( start1 );
                hipEventDestroy( stop1 );

//                cudaMemcpy(C,Cd,dimension*dimension*sizeof(float),cudaMemcpyDeviceToHost);

        free(A);
        free(B);
        free(C);
        hipFree(Ad);
        hipFree(Bd);
        hipFree(Cd);

        return 0;
}


