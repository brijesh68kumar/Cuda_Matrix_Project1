
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

/* change dimension size as needed */
const int dimension = 1024 ;
const int blocksize = 20;
const int K = 16;

__global__ void gpuMM(float *A, float *B, float *C,int N)
{
	
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	float sum = 0;
	for (int n = 0; n < N; ++n)
	    sum =sum+ A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
	for (int n = 0; n < N; ++n)
	printf("%d\t",C[row*dimension+col]);
}


int main(int argc, char *argv[])
{
		hipEvent_t start1, stop1;
		float time;

		hipEventCreate(&start1);
		hipEventCreate(&stop1);
        
		int i, j;
        float *A, *B, *C;// start, end;
		float *Ad, *Bd, *Cd;

        A = (float*)malloc(dimension*dimension*sizeof(float));
        B = (float*)malloc(dimension*dimension*sizeof(float));
        C = (float*)malloc(dimension*dimension*sizeof(float));		
		
        srand(292);

        for(i = 0; i < dimension; i++)
                for(j = 0; j < dimension; j++)
                {   
                        A[dimension*i+j] = 1;
                        B[dimension*i+j] = 2;
                       // C[dimension*i+j] = 0.0;
                }   
				
		hipMalloc( (void**)&Ad, dimension*dimension*sizeof(float) );
		hipMemcpy( Ad, A, dimension*dimension*sizeof(float), hipMemcpyDefault );

		hipMalloc( (void**)&Bd, dimension*dimension*sizeof(float) );
		hipMemcpy( Bd, B, dimension*dimension*sizeof(float), hipMemcpyDefault );
		
		hipMalloc( (void**)&Cd, dimension*dimension*sizeof(float) );

		dim3 threadBlock(blocksize,blocksize);
		dim3 grid(K,K);
		
        //start = timestamp();
		hipEventRecord( start1, 0 );

		gpuMM<<<grid,threadBlock>>>( Ad,Bd,Cd,dimension);
		hipMemcpy( Cd, C, dimension*dimension*sizeof(float), hipMemcpyDefault );

        //end = timestamp();

		hipEventRecord( stop1, 0 );
		hipEventSynchronize( stop1 );
		
		hipEventElapsedTime( &time, start1, stop1 );
		
		//print(C,N);
		printf("\nTime to calculate results on GPU: %f ms.\n", time);
		hipEventDestroy( start1 );
		hipEventDestroy( stop1 );
        

        free(A);
        free(B);
        free(C);
		hipFree(Ad);
		hipFree(Bd);
		hipFree(Cd);

        return 0;
}
